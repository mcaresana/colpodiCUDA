#include <iostream>
#include <fstream>
#include <string>
#include "Seed.h"
#include "MarketData.h"
#include "OptionData.h"
#include "GPUData.h"
#include "SimulationParameters.h"

using namespace std;

void Reader(MarketData &MarketInput, OptionData &OptionInput, GPUData &GPUInput, SimulationParameters &Parameters){

    string InputFile="DATA/input.conf";
    cout<<"Lettura file di input: "<<InputFile<<" ..."<<endl;
    fstream file;
    file.open(InputFile.c_str() , ios::in);

    if(file.fail()){
        cout<< "ERRORE: file di configurazione non trovato. "<<  endl;
    }
    string temp, word;
    int Threads=0, Streams=0, BlockSize=0;
    int EulerApproximation=0, OptionType=0;
    double Volatility=0, Drift=0;
    double InitialPrice=0, MaturityDate=0, StrikePrice=0;
    int DatesToSimulate=0, EulerSubStep=1;
    double ParamK=0, ParamB=0, ParamN=0;

    while (!file.eof()){
        file>>word;
        if (word=="THREADS") {
            file>> temp;
            Threads=atoi(temp.c_str());
        }
        if (word=="STREAMS"){
            file>> temp;
            Streams=atoi(temp.c_str());
        }
        if (word=="BLOCK_SIZE"){
            file>> temp;
            BlockSize=atoi(temp.c_str());
        }
        if (word=="EULER_APPROX"){
            file>> temp;
            EulerApproximation=atoi(temp.c_str());
        }
        if (word=="OPTION_TYPE"){
            file>> temp;
            OptionType=atoi(temp.c_str());
        }
        if (word=="VOLATILITY"){
            file>> temp;
            Volatility=atof(temp.c_str());
        }
        if (word=="DRIFT"){
            file>> temp;
            Drift=atof(temp.c_str());
        }
        if (word=="INITIAL_PRICE"){
            file>> temp;
            InitialPrice=atof(temp.c_str());
        }
        if (word=="MATURITY_DATE"){
            file>> temp;
            MaturityDate=atof(temp.c_str());
        }
        if (word=="DATES_TO_SIMULATE"){
            file>> temp;
            DatesToSimulate=atoi(temp.c_str());
        }
        if (word=="STRIKE_PRICE"){
            file>> temp;
            StrikePrice=atof(temp.c_str());
        }
        if (word=="PARAMETER_K"){
            file>> temp;
            ParamK=atof(temp.c_str());
        }
        if (word=="PARAMETER_N"){
            file>> temp;
            ParamN=atof(temp.c_str());
        }
        if (word=="PARAMETER_B"){
            file>> temp;
            ParamB=atof(temp.c_str());
        }
        if (word=="EULER_SUB_STEP"){
            file>> temp;
            EulerSubStep=atof(temp.c_str());
        }
    }

    file.close();

    GPUInput.Threads=Threads;
    GPUInput.Streams=Streams;
    GPUInput.BlockSize=BlockSize;

    bool EulerBool=EulerApproximation;
    Parameters.EulerApprox=EulerBool;
    Parameters.OptionType=OptionType;

    MarketInput.Volatility=Volatility;
    MarketInput.Drift=Drift;
    MarketInput.EquityInitialPrice=InitialPrice;

    if(EulerBool==false)
        EulerSubStep=1;

    OptionInput.MaturityDate=MaturityDate;
    OptionInput.NumberOfDatesToSimulate=DatesToSimulate,
    OptionInput.EulerSubStep=EulerSubStep;
    OptionInput.StrikePrice=StrikePrice;
    OptionInput.OptionTypeCallOrPut=OptionType;
    OptionInput.B=ParamB;
    OptionInput.N=ParamN;
    OptionInput.K=ParamK;

    /*
    if(OptionType==0){
        *OptionInput = new OptionDataForward(MaturityDate, DatesToSimulate, EulerSubStep);
    }
    if(OptionType==1 || OptionType==2){
        *OptionInput = new OptionDataPlainVanilla(MaturityDate, DatesToSimulate, EulerSubStep, StrikePrice, OptionType);
    }
    if(OptionType==3){
        *OptionInput = new OptionDataAbsolutePerformanceBarrier(MaturityDate, DatesToSimulate, EulerSubStep, ParamB, ParamN, ParamK);
    }
    */
}

void GetSeeds(Seed* SeedVector, int THREADS){

    srand(17*17);

    for(int i=0; i<THREADS; i++){
        SeedVector[i].S1=rand()%(UINT_MAX-128)+128;
        SeedVector[i].S2=rand()%(UINT_MAX-128)+128;
        SeedVector[i].S3=rand()%(UINT_MAX-128)+128;
        SeedVector[i].S4=rand();
    }
};

void MemoryAllocationGPUandCPU(Statistics** PayOffsGPU, Statistics** PayOffsCPU, Seed** SeedVector, Statistics** _PayOffsGPU, Seed** _SeedVector, size_t sizeSeedVector, size_t sizeDevStVector, int THREADS){
    cout<<"Allocazione della memoria..."<<endl;
    *PayOffsGPU=new Statistics[THREADS];
    *PayOffsCPU = new Statistics[THREADS];
    *SeedVector= new Seed[THREADS];

    hipMalloc((void**)& *_PayOffsGPU, sizeDevStVector);
    hipMalloc((void**)& *_SeedVector, sizeSeedVector);
};

void MemoryAllocationGPU(Statistics** PayOffsGPU, Seed** SeedVector, Statistics** _PayOffsGPU, Seed** _SeedVector, size_t sizeSeedVector, size_t sizeDevStVector, int THREADS){
    cout<<"Allocazione della memoria..."<<endl;
    *PayOffsGPU=new Statistics[THREADS];
    *SeedVector= new Seed[THREADS];

    hipMalloc((void**)& *_PayOffsGPU, sizeDevStVector);
    hipMalloc((void**)& *_SeedVector, sizeSeedVector);
};

void MemoryDeallocationGPUandCPU(Statistics* PayOffsGPU, Statistics* PayOffsCPU, Seed* SeedVector, Statistics* _PayOffsGPU, Seed* _SeedVector){
    delete[] PayOffsGPU;
    delete[] PayOffsCPU;
    delete[] SeedVector;

    hipFree(_PayOffsGPU);
    hipFree(_SeedVector);

    hipDeviceReset();
};

void MemoryDeallocationGPU(Statistics* PayOffsGPU, Seed* SeedVector, Statistics* _PayOffsGPU, Seed* _SeedVector){
    delete[] PayOffsGPU;
    delete[] SeedVector;

    hipFree(_PayOffsGPU);
    hipFree(_SeedVector);

    hipDeviceReset();
};
