#include "hip/hip_runtime.h"
#ifndef _KernelFunctions_cu_
#define _KernelFunctions_cu_

#include "MonteCarloPricer.h"
#include "Statistics.h"
#include "RandomGenerator.h"
#include "RandomGeneratorCombined.h"
#include "Seed.h"
#include "MarketData.h"
#include "OptionData.h"
#include "SimulationParameters.h"
#include "Option.h"

/*############################ Kernel Functions ############################*/

__host__ __device__ void TrueKernel(Seed* SeedVector, Statistics* PayOffs, int streams, MarketData MarketInput, OptionDataContainer OptionInput, SimulationParameters Parameters, int cont){

    RandomGenerator* Generator= new RandomGeneratorCombined(SeedVector[cont], false);

    StocasticProcess* Process;
    if(Parameters.EulerApprox==false)
        Process=new ExactLogNormalProcess(Generator);
    if(Parameters.EulerApprox==true)
        Process=new EulerLogNormalProcess(Generator);

    MontecarloPath* Path=new MontecarloPath(MarketInput, OptionInput.MaturityDate, OptionInput.NumberOfFixingDate, Process, Parameters.EulerSubStep);

    OptionData OptionParameters;
    OptionParameters.MaturityDate=OptionInput.MaturityDate;
    OptionParameters.NumberOfFixingDate=OptionInput.NumberOfFixingDate;
    OptionParameters.OptionType=OptionInput.OptionType;

    Option* Option;
    if( OptionInput.OptionType==0){
        Option=new OptionForward(OptionParameters, Path);
    }
    if( OptionInput.OptionType==1 || OptionInput.OptionType==2){
        OptionParameters.AdditionalParameters=new double[1];
        OptionParameters.AdditionalParameters[0]=OptionInput.StrikePrice;

        Option=new OptionPlainVanilla(OptionParameters, Path);
    }

    if( OptionInput.OptionType==3){
        OptionParameters.AdditionalParameters=new double[3];
        OptionParameters.AdditionalParameters[0]=OptionInput.B;
        OptionParameters.AdditionalParameters[1]=OptionInput.K;
        OptionParameters.AdditionalParameters[2]=OptionInput.N;

        Option=new OptionAbsolutePerformanceBarrier(OptionParameters, Path);
    }

    MonteCarloPricer Pricer(Option, streams);

    PayOffs[cont].Reset();
    Pricer.ComputePrice(&PayOffs[cont]);

}

__global__ void Kernel(Seed* SeedVector, Statistics* PayOffs, int streams, MarketData MarketInput, OptionDataContainer OptionInput, SimulationParameters Parameters){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    TrueKernel(SeedVector,PayOffs, streams, MarketInput, OptionInput, Parameters, i);
};

//## Funzione che gira su CPU che restituisce due vettori con sommme dei PayOff e dei PayOff quadrati. ##

__host__ void KernelSimulator(Seed* SeedVector, Statistics* PayOffs, int streams, MarketData MarketInput, OptionDataContainer OptionInput, SimulationParameters Parameters, int threads){

    for(int i=0; i<threads; i++) TrueKernel(SeedVector, PayOffs, streams, MarketInput, OptionInput, Parameters, i);

};

#endif
