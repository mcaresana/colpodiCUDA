#include "hip/hip_runtime.h"
#include <cmath>
#include "StochasticProcess.h"

__host__ __device__ ExactLogNormalProcess::ExactLogNormalProcess(RandomGenerator* Generator){
    _Generator=Generator;
};
__host__ __device__ RandomGenerator* StochasticProcess::GetRandomGenerator(){
    return _Generator;
};
__host__ __device__ void ExactLogNormalProcess::Step(UnderlyingPrice * Step, double TimeStep, double RandomNumber){
    double Drift=Step->Anagraphy->Drift;
    double Volatility=Step->Anagraphy->Volatility;
    Step->Price=Step->Price*exp((Drift - (Volatility*Volatility)/2)*TimeStep + Volatility*sqrt(TimeStep)*RandomNumber);

};


__host__ __device__  EulerLogNormalProcess::EulerLogNormalProcess(RandomGenerator* Generator){
    _Generator=Generator;
};

__host__ __device__ void EulerLogNormalProcess::Step(UnderlyingPrice * Step, double TimeStep,  double RandomNumber){
    double Drift=Step->Anagraphy->Drift;
    double Volatility=Step->Anagraphy->Volatility;
    Step->Price=Step->Price*(1.+Drift*TimeStep+Volatility*sqrt(TimeStep)*RandomNumber);
};
