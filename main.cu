#include "hip/hip_runtime.h"
/*##############################################################################################################################################################
# Pricer MonteCarlo di opzioni la cui dinamica e' determinata da processi lognormali esatti o approssimati.                                                    #
#                                                                                                                                                              #
# Usage: ./Pricer                                                                                                                                              #
# Speficicare: Dati di input del processo (MarketData), Dati di input dell'opzione (OptionData), tipo di Pay Off (guarda in PayOff.h per quelli implementati), #
#             tipo di processo (guarda in StocasticProcess.h per quelli implementati).                                                                         #
#                                                                                                                                                              #
# Output: Prezzo stimato secondo il Pay Off specificato e corrispondente errore MonteCarlo.                                                                    #
##############################################################################################################################################################*/

#include <iostream>
#include <cstdio>
#include <ctime>
#include "MonteCarloPricer.h"
#include "Statistics.h"
#include "DataTypes.h"
#include "KernelFunctions.cu"
#include "Utilities.cu"

using namespace std;

int main(){

//## Inizializzazione parametri di mercato e opzione. ##########################

    int THREADS;
    int STREAMS;

    MarketData MarketInput;
    OptionData OptionInput;

    Reader(MarketInput, OptionInput, THREADS, STREAMS);

//## Allocazione di memoria. ###################################################

    double *PayOffsGPU = new double[THREADS];
    double *PayOffs2GPU = new double[THREADS];
    double *PayOffsCPU = new double[THREADS];
    double *PayOffs2CPU = new double[THREADS];
    Seed *SeedVector= new Seed[THREADS];

    double *_PayOffsGPU;
    double *_PayOffs2GPU;
    Seed *_SeedVector;

    size_t sizeSeedVector = THREADS * sizeof(Seed);
    size_t sizePO = THREADS * sizeof(double);

    hipMalloc((void**)& _PayOffsGPU, sizePO);
    hipMalloc((void**)& _PayOffs2GPU, sizePO);
    hipMalloc((void**)& _SeedVector, sizeSeedVector);

//## Costruzione vettore dei seed. #############################################

    GetSeeds(SeedVector, THREADS);

    hipMemcpy(_SeedVector, SeedVector, sizeSeedVector, hipMemcpyHostToDevice);

//## Calcolo dei PayOff su GPU. ################################################

    int blockSize=512;
    int gridSize = (THREADS + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    Kernel<<<gridSize, blockSize>>>(_SeedVector, _PayOffsGPU, _PayOffs2GPU, STREAMS, MarketInput, OptionInput);
    hipEventRecord(stop);

    hipMemcpy(PayOffsGPU, _PayOffsGPU, sizePO, hipMemcpyDeviceToHost);
    hipMemcpy(PayOffs2GPU, _PayOffs2GPU, sizePO, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

//## Calcolo dei PayOff su CPU. ################################################

    clock_t startcpu;
    double duration;

    startcpu = clock();
    KernelSimulator(SeedVector, PayOffsCPU, PayOffs2CPU, STREAMS, MarketInput, OptionInput, THREADS);
    duration = (clock() - startcpu ) / (double) CLOCKS_PER_SEC;

//## Calcolo PayOff ed errore monte carlo a partire dai valori di PayOff simulati. ##

    Statistics OptionGPU(PayOffsGPU, PayOffs2GPU, THREADS, STREAMS);
    Statistics OptionCPU(PayOffsCPU, PayOffs2CPU, THREADS, STREAMS);

//## Stampa su file dei valori. ##############################################

    cout<<"Valori GPU"<<endl;
    OptionGPU.Print();
    OptionGPU.Print("DATA/outputGPU.dat");
    cout<<"Tempo di calcolo: "<<milliseconds<<" ms"<<endl<<endl;

    cout<<"Valori CPU"<<endl;
    OptionCPU.Print();
    OptionCPU.Print("DATA/outputCPU.dat");
    cout<<"Tempo di calcolo: "<<duration*1000<<" ms"<<endl;


//## Liberazione memoria. ######################################################

    hipFree(_PayOffsGPU);
    hipFree(_PayOffs2GPU);
    hipFree(_SeedVector);

    delete[] PayOffsGPU;
    delete[] PayOffsCPU;
    delete[] PayOffs2GPU;
    delete[] PayOffs2CPU;
    delete[] SeedVector;

    return 0;
}
