#include "hip/hip_runtime.h"
#include <iostream>
#include "Pricer.h"
#include "Statistics.h"
#include "DataTypes.h"

using namespace std;

__global__ void Kernel(Seed* S, float* PayOffs, float* PayOffs2, int streams){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
//    int threads = blockDim.x * gridDim.x;

    MarketData MarketInput;
    MarketInput.Volatility=0.3;
    MarketInput.Drift=0.04;
    MarketInput.SInitial=100;

    OptionData OptionInput;
    OptionInput.TInitial=0;
    OptionInput.TFinal=1;
    OptionInput.NSteps=12;
    OptionInput.StrikePrice=100;

    MonteCarloPricer P(MarketInput, OptionInput, streams, S[i]);
    P.GetPrice();

    float SumPayOff=P.GetPayOff();
    float SumPayOff2=P.GetPayOff2();

    PayOffs[i]=SumPayOff;
    PayOffs2[i]=SumPayOff2;
}

__host__ void KernelSimulator(Seed* S, float* PayOffs, float* PayOffs2, int streams, int threads){

    MarketData MarketInput;
    MarketInput.Volatility=0.3;
    MarketInput.Drift=0.04;
    MarketInput.SInitial=100;


    OptionData OptionInput;
    OptionInput.TInitial=0;
    OptionInput.TFinal=1;
    OptionInput.NSteps=12;
    OptionInput.StrikePrice=100;

    for(int i=0; i<threads; i++){
        MonteCarloPricer P(MarketInput, OptionInput, streams, S[i]);
        P.GetPrice();
        float SumPayOff=P.GetPayOff();
        float SumPayOff2=P.GetPayOff2();

        PayOffs[i]=SumPayOff;
        PayOffs2[i]=SumPayOff2;
    }

}

int main(){
    int streams=100;
    int threads=1024;

    srand(657);

//    unsigned int *S = new unsigned int[4*threads];
    Seed *S= new Seed[threads];
    float *PayOffs = new float[threads];
    float *PayOffs2 = new float[threads];

    for(int i=0; i<threads; i++){
		S[i].S1=rand()+128;
        S[i].S2=rand()+128;
        S[i].S3=rand()+128;
        S[i].S4=rand()+128;
	}

// Cuda

    float *_PayOffs;
    float *_PayOffs2;
    Seed *_S;

    size_t sizeS = threads * sizeof(Seed);
    size_t sizePO = threads * sizeof(float);

    hipMalloc((void**)& _PayOffs, sizePO);
    hipMalloc((void**)& _PayOffs2, sizePO);
    hipMalloc((void**)& _S, sizeS);

    hipMemcpy(_S, S, sizeS, hipMemcpyHostToDevice);

    int blockSize=512;
    int gridSize = (threads + blockSize - 1) / blockSize;

    Kernel<<<gridSize, blockSize>>>(_S, _PayOffs, _PayOffs2, streams);


//    KernelSimulator(S, PayOffs, PayOffs2, streams, threads);

    hipMemcpy(PayOffs, _PayOffs, sizePO, hipMemcpyDeviceToHost);
    hipMemcpy(PayOffs2, _PayOffs2, sizePO, hipMemcpyDeviceToHost);

    Statistics Option(PayOffs, PayOffs2, threads, streams);

    cout<<"Prezzo: "<<Option.GetPrice()<<endl;
    cout<<"Errore MonteCarlo: "<<Option.GetMCError()<<endl;

    hipFree(_PayOffs);
    hipFree(_PayOffs2);
    hipFree(_S);

    delete[] PayOffs;
    delete[] PayOffs2;
    delete[] S;

    return 0;
}
