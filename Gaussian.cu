#include "hip/hip_runtime.h"
#include "Gaussian.h"
#include <cmath>

 __host__ __device__ Gaussian::Gaussian(bool BoxMullerWithReExtraction){
  _BoxMullerWithReExtraction=BoxMullerWithReExtraction;
  _Status=true;
}
 __host__ __device__ double Gaussian::GetGaussianRandomNumber(RandomGenerator* Generator){
  if(_Status==true){
      double u=Generator->GetUniformRandomNumber();
      double v=Generator->GetUniformRandomNumber();
      if(_BoxMullerWithReExtraction==false){
          if(u==0) return this->GetGaussianRandomNumber(Generator);
          _SavedRandomNumber=sqrt(-2.*log(u))*sin(2*M_PI*v);
          _Status=false;
          return sqrt(-2.*log(u))*cos(2*M_PI*v);
      }
      else{
            u=2*u-1;
            v=2*v-1;
            double r=u*u+v*v;
            if(r==0 || r>=1) return this->GetGaussianRandomNumber(Generator);
            _SavedRandomNumber=v*sqrt(-2.*log(r)/r);
            _Status=false;
            return u*sqrt(-2.*log(r)/r);
      }
  }
  else{
    _Status=true;
    return _SavedRandomNumber;
  }
};
